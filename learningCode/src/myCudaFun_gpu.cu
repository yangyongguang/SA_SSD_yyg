#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"

#include "myCudaFun_gpu.h"
#include ""

__global__ void add_kernel_fast(int n, const float * data1, const float *data2, float * data3) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id >= n) return;
    // printf("current idx: %d\n", id);
    data3[id] = data1[id] + data2[id];
    // printf("idx: %d, %f + %f = %f\n", id, data1[id], data2[id], data3[id]);
}

__global__ void run_on_gpu() {
	printf("GPU thread info X:%d Y:%d Z:%d\t block info X:%d Y:%d Z:%d\n",
		threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z);
}

// void add_kernerl_launcher_fast(int n, const float *data1, const float *data2, float *data3, hipStream_t stream) {
void add_kernerl_launcher_fast(int n, const float *data1, const float *data2, float *data3) {
    dim3 blocks(DIVUP(n, THREADS_PER_BLOCK));  // blockIdx.x(col),  blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    // printf("number of tensor size: %d\n", n);
    add_kernel_fast<<<blocks, threads>>>(n, data1, data2, data3);
    // run_on_gpu<<<blocks, threads, 0>>>();
    hipDeviceSynchronize();

    hipError_t err;
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}